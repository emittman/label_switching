#include "hip/hip_runtime.h"
#include<fns.h>
#include<thrust/device_vector.h>
#include<thrust/host_vector.h>
#include<thrust/copy.h>
#include<thrust/sequence.h>
#include<thrust/sort.h>
#include<hiprand.h>

typedef thrust::device_vector<double> fvec_d;
typedef thrust::device_vector<int> ivec_d;
typedef thrust::host_vector<int> ivec_d;

__global__ void setup_kernel(int seed, int n_threads, hiprandState *states) {
  
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if(id < n_threads){
  /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(seed, id, 0, &states[id]);
  }
}

__global__ void getExponential(hiprandState *states, int n_threads, double *weights, double *result){
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if(id < n_threads){
    result[id] = -log(hiprand_uniform(&(states[id]))) / weights[id];
  }
}

void sample_wwr(hiprandState *states, fvec_d &weights, ivec_d &result){
  unsigned N = weights.size();
  fvec_d e(N);
  thrust::sequence(result.begin(), result.end());
  unsigned blocksize = 512;
  unsigned nblocks = N/512 + 1;
  getExponential<<<nblocks, blocksize>>>(states, N, thrust::raw_pointer_cast(weights.data()), thrust::raw_pointer_cast(e.data()));
  thrust::sort_by_key(e.begin(), e.end(), result.begin());
}

extern "C" SEXP Rsample_wwr(SEXP Rseed, SEXP Rweights){
  int N = length(Rweights), seed = INTEGER(Rseed)[0];
  fvec_d weights(REAL(Rweights), REAL(Rweights) + N);
  ivec_d out_d(N);
  //instantiate RNGs
  hiprandState *devStates;
  CUDA_CALL(hipMalloc((void **) &devStates, N * sizeof(hiprandState)));
  unsigned blocksize = 512;
  unsigned nblocks = N/512 + 1;
  setup_kernel<<<nblocks, blocksize>>>(seed, N, devStates);
  
  sample_wwr(devStates, weights, out_d);
  
  thrust::host_vector<int> out_h(N);
  thrust::copy(out_d.begin(), out_d.end(), out_h.begin());
  SEXP out = PROTECT(allocVector(INTSXP, N));
  for(int i=0; i<N; i++) INTEGER(out)[i] = out_h[i];
  UNPROTECT(1);
  hipFree(devStates);
  return out;
}

